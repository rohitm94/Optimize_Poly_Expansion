
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define BLOCKSIZE 512

__global__ void polynomial_expansion(float *poly, int degree, int n, float *array)
{
    //TODO: Write code to use the GPU here!
    //code should write the output back to array
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        float temp = array[index];
        float out = 0, xtothepowerof = 1;
        for (int i = 0; i <= degree; i++)
        {
            out += xtothepowerof * poly[i];
            xtothepowerof *= temp;
        }
        array[index] = out;
    }
}

int main(int argc, char *argv[])
{
    //TODO: add usage

    if (argc < 3)
    {
        std::cerr << "usage: " << argv[0] << " n degree" << std::endl;
        return -1;
    }

    char *ptr;
    long long int n = strtol(argv[1],&ptr,10);
    int degree = atoi(argv[2]);
    int nbiter = 2;

    float* array = NULL;
    float* poly = NULL;

    hipHostMalloc((void **)&array,sizeof(float)*(n/2), hipHostMallocDefault);
    hipHostMalloc((void **)&poly,sizeof(float)*(degree+1), hipHostMallocDefault);
    for (int i = 0; i < n/2; ++i)
        array[i] = 1.;

    for (int i = 0; i < (degree + 1); ++i)
        poly[i] = 1.;

    float *d_array, *d_poly;

    hipMalloc((void **)&d_array, (n/2) * sizeof(float));
    hipMalloc((void **)&d_poly, (degree + 1) * sizeof(float));

    long long int size = n * sizeof(float) / 8;


    hipStream_t stream[4];
    for (int i = 0; i < 4; ++i){
        hipStreamCreate(&stream[i]);
    }



    hipMemcpyAsync(d_poly, poly, (degree + 1) * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    hipSetDevice(1);
    float* array1 = NULL;
    float* poly1 = NULL;

    hipHostMalloc((void **)&array1,sizeof(float)*(n/2), hipHostMallocDefault);
    hipHostMalloc((void **)&poly1,sizeof(float)*(degree+1), hipHostMallocDefault);
    for (int i = 0; i < n/2; ++i)
        array1[i] = 1.;

    for (int i = 0; i < (degree + 1); ++i)
        poly1[i] = 1.;

    float *d_array1, *d_poly1;

    hipMalloc((void **)&d_array1, (n/2) * sizeof(float));
    hipMalloc((void **)&d_poly1, (degree + 1) * sizeof(float));


    hipStream_t stream1[4];
    for (int i = 0; i < 4; ++i){
        hipStreamCreate(&stream1[i]);
    }



    hipMemcpyAsync(d_poly1, poly1, (degree + 1) * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();


        std::chrono::time_point<std::chrono::system_clock> begin, end;
        begin = std::chrono::system_clock::now();
        for(int k = 1; k <=nbiter; k++){
            hipSetDevice(0);
            for (int i = 0; i < 4; ++i) {
                hipMemcpyAsync(d_array+ i*size, array + i*size,size, hipMemcpyHostToDevice, stream[i]);
                }
            for (int i = 0; i < 4; ++i) {
                 polynomial_expansion <<<((n/8) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream[i]>>>(d_poly, degree, n/8, d_array + i*size);
                hipMemcpyAsync(array+ i*size, d_array+ i*size,size, hipMemcpyDeviceToHost, stream[i]);
                }
            hipSetDevice(1);
            for (int i = 0; i < 4; ++i) {
                hipMemcpyAsync(d_array1+ i*size, array1 + i*size,size, hipMemcpyHostToDevice, stream1[i]);
                }
            for (int i = 0; i < 4; ++i) {
                polynomial_expansion <<<((n/8) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream1[i]>>>(d_poly, degree, n/8, d_array1 + i*size);
                hipMemcpyAsync(array1+ i*size, d_array1+ i*size,size, hipMemcpyDeviceToHost, stream1[i]);
                }
            }

        hipDeviceSynchronize();
        end = std::chrono::system_clock::now();
        std::chrono::duration<double> totaltime = (end - begin);

        hipSetDevice(0);
        for (int i = 0; i < 4; ++i){
            hipStreamDestroy(stream[i]);
        }
        hipFree(d_array);
        hipFree(d_poly);
        hipSetDevice(1);
        for (int i = 0; i < 4; ++i){
            hipStreamDestroy(stream1[i]);
        }
        hipFree(d_array1);
        hipFree(d_poly1);

        double pciBW = 1.50e+10, gpumemBW = 2.88e+11 , gpuflopRate = 1.43e+12 , pciLat = 8.80594e-06;

        double HtD = pciLat + double(((nbiter*(n/2))*(sizeof(float)))/pciBW);
        std::cout<<"HTD: "<<HtD<<std::endl;
        double DtH = pciLat + double(((nbiter*(n/2))*(sizeof(float)))/pciBW);

        double dProc = std::max(double((3.0*(n/2)*(degree+1))/(gpuflopRate)),(double(sizeof(float)*((nbiter*(n/2))+degree+1))/(gpumemBW)));
        std::cout<<"dproc:"<<dProc<<" "<<double((3.0*(n/2)*(degree+1))/(gpuflopRate))<<" "<<(double(sizeof(float)*((nbiter*(n/2))+degree+1))/(gpumemBW))<<std::endl;
        double ideal_time = std::max(dProc,2*(HtD+DtH));

        std::cout << double(n*sizeof(float))<< " " << degree << " " << ideal_time << " " << totaltime.count() << " " << (double(n*(degree+1)))/(ideal_time) << " " << ((n)*nbiter)/totaltime.count() << std::endl;
        hipSetDevice(0);
        hipHostFree(array);
        hipHostFree(poly);
        hipSetDevice(1);
        hipHostFree(array1);
        hipHostFree(poly1);

        return 0;
    }

