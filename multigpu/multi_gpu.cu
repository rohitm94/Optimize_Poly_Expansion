
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define BLOCKSIZE 512

__global__ void polynomial_expansion(float *poly, int degree, int n, float *array)
{
    //TODO: Write code to use the GPU here!
    //code should write the output back to array
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        float temp = array[index];
        float out = 0, xtothepowerof = 1;
        for (int i = 0; i <= degree; i++)
        {
            out += xtothepowerof * poly[i];
            xtothepowerof *= temp;
        }
        array[index] = out;
    }
}

int main(int argc, char *argv[])
{
    //TODO: add usage

    if (argc < 3)
    {
        std::cerr << "usage: " << argv[0] << " n degree" << std::endl;
        return -1;
    }

    char *ptr;
    long long int n = strtol(argv[1],&ptr,10);
    int degree = atoi(argv[2]);
    int nbiter = 2;

    float* array = NULL;
    float* poly = NULL;

    hipHostMalloc((void **)&array,sizeof(float)*(n/2), hipHostMallocDefault);
    hipHostMalloc((void **)&poly,sizeof(float)*(degree+1), hipHostMallocDefault);
    for (int i = 0; i < n/2; ++i)
        array[i] = 1.;

    for (int i = 0; i < (degree + 1); ++i)
        poly[i] = 1.;

    float *d_array, *d_poly;

    hipMalloc((void **)&d_array, (n/2) * sizeof(float));
    hipMalloc((void **)&d_poly, (degree + 1) * sizeof(float));

    long long int size = n * sizeof(float) / 8;


    /*cudaStream_t stream[4];
    for (int i = 0; i < 4; ++i){
        cudaStreamCreate(&stream[i]);
    }*/



    hipMemcpy(d_poly, poly, (degree + 1) * sizeof(float), hipMemcpyHostToDevice);

    hipSetDevice(1);
    float* array1 = NULL;
    float* poly1 = NULL;

    hipHostMalloc((void **)&array1,sizeof(float)*(n/2), hipHostMallocDefault);
    hipHostMalloc((void **)&poly1,sizeof(float)*(degree+1), hipHostMallocDefault);
    for (int i = 0; i < n/2; ++i)
        array1[i] = 1.;

    for (int i = 0; i < (degree + 1); ++i)
        poly1[i] = 1.;

    float *d_array1, *d_poly1;

    hipMalloc((void **)&d_array1, (n/2) * sizeof(float));
    hipMalloc((void **)&d_poly1, (degree + 1) * sizeof(float));


    /*cudaStream_t stream1[4];
    for (int i = 0; i < 4; ++i){
        cudaStreamCreate(&stream1[i]);
    }*/



    hipMemcpy(d_poly1, poly1, (degree + 1) * sizeof(float), hipMemcpyHostToDevice);


        std::chrono::time_point<std::chrono::system_clock> begin, end;
        begin = std::chrono::system_clock::now();
        for(int k = 1; k <=nbiter; k++){
            hipSetDevice(0);
            hipStream_t stream[4];
            for (int i = 0; i < 4; ++i){
                hipStreamCreate(&stream[i]);
            }
            //for (int i = 0; i < 4; ++i) {
                hipMemcpyAsync(d_array+ 0*size, array + 0*size,size, hipMemcpyHostToDevice, stream[0]);
                polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream[0]>>>(d_poly, degree, n/4, d_array + 0*size);
                hipMemcpyAsync(array+ 0*size, d_array+ 0*size,size, hipMemcpyDeviceToHost, stream[0]);

                hipMemcpyAsync(d_array+ 1*size, array + 1*size,size, hipMemcpyHostToDevice, stream[1]);
                polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream[1]>>>(d_poly, degree, n/4, d_array + 1*size);
                hipMemcpyAsync(array+ 1*size, d_array+ 1*size,size, hipMemcpyDeviceToHost, stream[1]);

                hipMemcpyAsync(d_array+ 2*size, array + 2*size,size, hipMemcpyHostToDevice, stream[2]);
                polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream[2]>>>(d_poly, degree, n/4, d_array + 2*size);
                hipMemcpyAsync(array+ 2*size, d_array+ 2*size,size, hipMemcpyDeviceToHost, stream[2]);

                hipMemcpyAsync(d_array+ 3*size, array + 3*size,size, hipMemcpyHostToDevice, stream[3]);
                polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream[3]>>>(d_poly, degree, n/4, d_array + 3*size);
                hipMemcpyAsync(array+ 3*size, d_array+ 3*size,size, hipMemcpyDeviceToHost, stream[3]);
                //}

            hipSetDevice(1);
            hipStream_t stream1[4];
            for (int i = 0; i < 4; ++i){
                hipStreamCreate(&stream1[i]);
            }
            hipMemcpyAsync(d_array1+ 0*size, array1 + 0*size,size, hipMemcpyHostToDevice, stream1[0]);
            polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream1[0]>>>(d_poly1, degree, n/4, d_array1 + 0*size);
            hipMemcpyAsync(array1+ 0*size, d_array1+ 0*size,size, hipMemcpyDeviceToHost, stream1[0]);

            hipMemcpyAsync(d_array1+ 1*size, array1 + 1*size,size, hipMemcpyHostToDevice, stream1[1]);
            polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream1[1]>>>(d_poly1, degree, n/4, d_array1 + 1*size);
            hipMemcpyAsync(array1+ 1*size, d_array1+ 1*size,size, hipMemcpyDeviceToHost, stream1[1]);

            hipMemcpyAsync(d_array1+ 2*size, array1 + 2*size,size, hipMemcpyHostToDevice, stream1[2]);
            polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream1[2]>>>(d_poly1, degree, n/4, d_array1 + 2*size);
            hipMemcpyAsync(array1+ 2*size, d_array1+ 2*size,size, hipMemcpyDeviceToHost, stream1[2]);

            hipMemcpyAsync(d_array1 + 3*size, array1 + 3*size,size, hipMemcpyHostToDevice, stream1[3]);
            polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream1[3]>>>(d_poly1, degree, n/4, d_array1 + 3*size);
            hipMemcpyAsync(array1 + 3*size, d_array1 + 3*size,size, hipMemcpyDeviceToHost, stream1[3]);
            //}
            hipStreamSynchronize(stream1[0]); 
            hipStreamSynchronize(stream1[1]);
            hipStreamSynchronize(stream1[2]);
            hipStreamSynchronize(stream1[3]);
            for (int i = 0; i < 4; ++i){
                hipStreamDestroy(stream1[i]);
            }
            hipSetDevice(0);
            hipStreamSynchronize(stream[0]); 
            hipStreamSynchronize(stream[1]);
            hipStreamSynchronize(stream[2]);
            hipStreamSynchronize(stream[3]);
            for (int i = 0; i < 4; ++i){
                hipStreamDestroy(stream[i]);
            }
            }

        hipDeviceSynchronize();
        end = std::chrono::system_clock::now();
        std::chrono::duration<double> totaltime = (end - begin);

        /*cudaSetDevice(0);
        for (int i = 0; i < 4; ++i){
            cudaStreamDestroy(stream[i]);
        }*/
        hipSetDevice(0);
        hipHostFree(array);
        hipHostFree(poly);
        hipFree(d_array);
        hipFree(d_poly);
        /*cudaSetDevice(1);
        for (int i = 0; i < 4; ++i){
            cudaStreamDestroy(stream1[i]);
        }*/
        hipSetDevice(1);
        hipHostFree(array1);
        hipHostFree(poly1);
        hipFree(d_array1);
        hipFree(d_poly1);

        double pciBW = 1.50e+10, gpumemBW = 2.88e+11 , gpuflopRate = 1.73e+12 , pciLat = 8.80594e-06;

        double HtD = double(((nbiter*(n/2))*(sizeof(float)))/pciBW);
        //std::cout<<"HTD: "<<HtD<<std::endl;
        double DtH = double(((nbiter*(n/2))*(sizeof(float)))/pciBW);

        double dProc = std::max(double((3.0*(n/2)*(degree+1))/(gpuflopRate)),(double(sizeof(float)*((nbiter*(n/2))+degree+1))/(gpumemBW)));
        //std::cout<<"dproc:"<<dProc<<" "<<double((3.0*(n/2)*(degree+1))/(gpuflopRate))<<" "<<(double(sizeof(float)*((nbiter*(n/2))+degree+1))/(gpumemBW))<<std::endl;
        double ideal_time = std::max(dProc,(HtD+DtH));

        std::cout << double(n*sizeof(float))<< " " << degree << " " << ideal_time << " " << totaltime.count() << " " << (double(n)/(ideal_time)) << " " << ((n)*nbiter)/totaltime.count() << std::endl;
        

        return 0;
    }

