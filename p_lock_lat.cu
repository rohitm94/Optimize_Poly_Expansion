
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

int main(int argc, char *argv[])
{

    if (argc < 3)
    {
        std::cerr << "usage: " << argv[0] << " n degree" << std::endl;
        return -1;
    }

    int n = atoi(argv[1]); //TODO: atoi is an unsafe function
    int degree = atoi(argv[2]);
    int nbiter = atoi(argv[3]);

    float *array ;
    hipHostMalloc((void **)&array, n * sizeof(float), hipHostMallocDefault);
    float *poly = new float[degree + 1];
    for (int i = 0; i < n; ++i)
        array[i] = 1.;

    for (int i = 0; i < degree + 1; ++i)
        poly[i] = 1.;

    float *d_array;

    hipMalloc((void **)&d_array, n * sizeof(float));

    std::chrono::time_point<std::chrono::system_clock> begin, end;
    begin = std::chrono::system_clock::now();

    for (int iter = 0; iter < nbiter; ++iter)
        hipMemcpy(d_array, array, n * sizeof(float), hipMemcpyHostToDevice);

    end = std::chrono::system_clock::now();
    std::chrono::duration<double> totaltime = (end - begin);

    hipFree(d_array);
    hipHostFree(array);

    std::cout << "Latency of "<< nbiter <<"times is " << totaltime.count() << std::endl;
    std::cout << "Latency of PCI express is " << totaltime.count() / nbiter << std::endl;

    delete[] array;
    delete[] poly;

    return 0;
}