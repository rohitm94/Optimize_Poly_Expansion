
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define BLOCKSIZE 256

__global__ void polynomial_expansion(float *poly, int degree, int n, float *array)
{
    //TODO: Write code to use the GPU here!
    //code should write the output back to array
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        float temp = array[index];
        float out = 0, xtothepowerof = 1;
        for (int i = 0; i <= degree; i++)
        {
            out += xtothepowerof * poly[i];
            xtothepowerof *= temp;
        }
        array[index] = out;
    }
}

int main(int argc, char *argv[])
{
    //TODO: add usage

    if (argc < 3)
    {
        std::cerr << "usage: " << argv[0] << " n degree" << std::endl;
        return -1;
    }

    int n = atoi(argv[1]); //TODO: atoi is an unsafe function
    int degree = atoi(argv[2]);
    int nbiter = 1;

    float* array = NULL;
    float* poly = NULL;
    
    hipHostMalloc((void **)&array,sizeof(float)*n, hipHostMallocDefault);
    hipHostMalloc((void **)&poly,sizeof(float)*(degree+1), hipHostMallocDefault);
    for (int i = 0; i < n; ++i)
        array[i] = 1.;

    for (int i = 0; i < degree + 1; ++i)
        poly[i] = 1.;

    float *d_array, *d_poly;

    hipMalloc((void **)&d_array, n * sizeof(float));
    hipMalloc((void **)&d_poly, (degree + 1) * sizeof(float));

    int size = n * sizeof(float) / 1;

    hipStream_t stream[1];
    for (int i = 0; i < 1; ++i)
        hipStreamCreate(&stream[i]);

    std::chrono::time_point<std::chrono::system_clock> begin, end;
    begin = std::chrono::system_clock::now();

    for (int i = 0; i < 1; ++i) {
        hipMemcpyAsync(d_array, array ,size, hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(d_poly, poly, (degree + 1) * sizeof(float), hipMemcpyHostToDevice, stream[i]);
        polynomial_expansion <<<(n + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream[i]>>>(d_poly, degree, n, d_array);
        hipMemcpyAsync(array, d_array,size, hipMemcpyDeviceToHost, stream[i]);
        }
    /*cudaMemcpy(d_array, array, n * sizeof(float), cudaMemcpyHostToDevice);
    cudaMemcpy(d_poly, poly, (degree + 1) * sizeof(float), cudaMemcpyHostToDevice);

    polynomial_expansion<<<(n + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_poly, degree, n, d_array);*/

    hipDeviceSynchronize();
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> totaltime = (end - begin);
    //cudaMemcpy(array, d_array, n * sizeof(float), cudaMemcpyDeviceToHost);

    hipFree(d_array);
    hipFree(d_poly);

    std::cerr << array[0] << std::endl;
    std::cout << n*sizeof(float)/1000 << " " << totaltime.count() << " " << ((n+degree+1)*sizeof(float)*nbiter)/totaltime.count() << std::endl;

    delete[] array;
    delete[] poly;

    return 0;
}