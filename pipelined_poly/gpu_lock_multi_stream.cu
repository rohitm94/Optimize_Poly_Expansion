
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define BLOCKSIZE 256

__global__ void polynomial_expansion(float *poly, int degree, int n, float *array)
{
    //TODO: Write code to use the GPU here!
    //code should write the output back to array
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        float temp = array[index];
        float out = 0, xtothepowerof = 1;
        for (int i = 0; i <= degree; i++)
        {
            out += xtothepowerof * poly[i];
            xtothepowerof *= temp;
        }
        array[index] = out;
    }
}

int main(int argc, char *argv[])
{
    //TODO: add usage

    if (argc < 3)
    {
        std::cerr << "usage: " << argv[0] << " n degree" << std::endl;
        return -1;
    }

    int n = atoi(argv[1]); //TODO: atoi is an unsafe function
    int degree = atoi(argv[2]);
    int nbiter = 1;

    float* array = NULL;
    float* poly = NULL;
    
    hipHostMalloc((void **)&array,sizeof(float)*n, hipHostMallocDefault);
    hipHostMalloc((void **)&poly,sizeof(float)*(degree+1), hipHostMallocDefault);
    for (int i = 0; i < n; ++i)
        array[i] = 1.;

    for (int i = 0; i < degree + 1; ++i)
        poly[i] = 1.;

    float *d_array, *d_poly;

    hipMalloc((void **)&d_array, n * sizeof(float));
    hipMalloc((void **)&d_poly, (degree + 1) * sizeof(float));

    int size = n * sizeof(float) / 4;

    std::cerr << "0th check"<<std::endl;
    hipStream_t stream[4];
    std::cerr << "1st check"<<std::endl;
    for (int i = 0; i < 4; ++i){
        hipStreamCreate(&stream[i]);
        std::cerr << i+2<<"th check"<<std::endl;
    }


    std::cerr << "first check"<<std::endl;

    std::chrono::time_point<std::chrono::system_clock> begin, end;
    begin = std::chrono::system_clock::now();

    for (int i = 0; i < 4; ++i) {
        std::cerr << "sec check"<<std::endl;
        hipMemcpyAsync(d_array+ i*size, array + i*size,size, hipMemcpyHostToDevice, stream[i]);
        std::cerr << "thi check"<<std::endl;
        hipMemcpyAsync(d_poly, poly, (degree + 1) * sizeof(float), hipMemcpyHostToDevice, stream[i]);
        std::cerr << "for check"<<std::endl;
        polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream[i]>>>(d_poly, degree, n/4, d_array);
        std::cerr << "fif check"<<std::endl;
        hipMemcpyAsync(array+ i*size, d_array+ i*size,size, hipMemcpyDeviceToHost, stream[i]);
        std::cerr << "six check"<<std::endl;
        }
    /*cudaMemcpy(d_array, array, n * sizeof(float), cudaMemcpyHostToDevice);
    cudaMemcpy(d_poly, poly, (degree + 1) * sizeof(float), cudaMemcpyHostToDevice);

    polynomial_expansion<<<(n + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_poly, degree, n, d_array);*/

    hipDeviceSynchronize();
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> totaltime = (end - begin);
    //cudaMemcpy(array, d_array, n * sizeof(float), cudaMemcpyDeviceToHost);

    hipFree(d_array);
    hipFree(d_poly);

    std::cerr << "sev check"<<std::endl;

    std::cout << n*sizeof(float)/1000 << " " << totaltime.count() << " " << ((n+degree+1)*sizeof(float)*nbiter)/totaltime.count() << std::endl;

    delete[] array;
    delete[] poly;

    return 0;
}