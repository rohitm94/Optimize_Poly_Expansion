
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define BLOCKSIZE 256

__global__ void polynomial_expansion(float *poly, int degree, int n, float *array)
{
    //TODO: Write code to use the GPU here!
    //code should write the output back to array
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        float temp = array[index];
        float out = 0, xtothepowerof = 1;
        for (int i = 0; i <= degree; i++)
        {
            out += xtothepowerof * poly[i];
            xtothepowerof *= temp;
        }
        array[index] = out;
    }
}

int main(int argc, char *argv[])
{
    //TODO: add usage

    if (argc < 3)
    {
        std::cerr << "usage: " << argv[0] << " n degree" << std::endl;
        return -1;
    }

    char *ptr;
    long long int n = strtol(argv[1],&ptr,10);
    int degree = atoi(argv[2]);
    int nbiter = 1;

    float* array = NULL;
    float* poly = NULL;

    hipHostMalloc((void **)&array,sizeof(float)*n, hipHostMallocDefault);
    hipHostMalloc((void **)&poly,sizeof(float)*(degree+1), hipHostMallocDefault);
    for (int i = 0; i < n; ++i)
        array[i] = 1.;

    for (int i = 0; i < degree + 1; ++i)
        poly[i] = 1.;

    float *d_array, *d_poly;

    hipMalloc((void **)&d_array, n * sizeof(float));
    hipMalloc((void **)&d_poly, (degree + 1) * sizeof(float));

    long long int size = n * sizeof(float) / 4;




    hipMemcpy(d_poly, poly, (degree + 1) * sizeof(float), hipMemcpyHostToDevice);

        std::chrono::time_point<std::chrono::system_clock> begin, end;
        begin = std::chrono::system_clock::now();
        for(int k = 1; k <=nbiter; k++){
            hipStream_t stream[4];
            for (int i = 0; i < 4; ++i){
                hipStreamCreate(&stream[i]);
            }
            //for (int i = 0; i < 4; ++i) {
                hipMemcpyAsync(d_array+ 0*size, array + 0*size,size, hipMemcpyHostToDevice, stream[0]);
                polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream[0]>>>(d_poly, degree, n/4, d_array + 0*size);
                hipMemcpyAsync(array+ 0*size, d_array+ 0*size,size, hipMemcpyDeviceToHost, stream[0]);

                hipMemcpyAsync(d_array+ 1*size, array + 1*size,size, hipMemcpyHostToDevice, stream[1]);
                polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream[1]>>>(d_poly, degree, n/4, d_array + 1*size);
                hipMemcpyAsync(array+ 1*size, d_array+ 1*size,size, hipMemcpyDeviceToHost, stream[1]);

                hipMemcpyAsync(d_array+ 2*size, array + 2*size,size, hipMemcpyHostToDevice, stream[2]);
                polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream[2]>>>(d_poly, degree, n/4, d_array + 2*size);
                hipMemcpyAsync(array+ 2*size, d_array+ 2*size,size, hipMemcpyDeviceToHost, stream[2]);

                hipMemcpyAsync(d_array+ 3*size, array + 3*size,size, hipMemcpyHostToDevice, stream[3]);
                polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream[3]>>>(d_poly, degree, n/4, d_array + 3*size);
                hipMemcpyAsync(array+ 3*size, d_array+ 3*size,size, hipMemcpyDeviceToHost, stream[3]);
                //}
                hipStreamSynchronize(stream[0]); 
                hipStreamSynchronize(stream[1]);
                hipStreamSynchronize(stream[2]);
                hipStreamSynchronize(stream[3]);
            for (int i = 0; i < 4; ++i){
                hipStreamDestroy(stream[i]);
            }
            }

        //cudaDeviceSynchronize();
        end = std::chrono::system_clock::now();
        std::chrono::duration<double> totaltime = (end - begin);


        hipFree(d_array);
        hipFree(d_poly);

        double pciBW = 1.50e+10, gpumemBW = 2.88e+11 , gpuflopRate = 1.43e+12 , pciLat = 8.80594e-06;

        double HtD =  double(((nbiter*n)*(sizeof(float)))/pciBW);
        double DtH =  double(((nbiter*n)*(sizeof(float)))/pciBW);

        double dProc = std::max(double((3.0*(n)*(degree+1))/(gpuflopRate)),(double(sizeof(float)*((nbiter*n)+degree+1))/(gpumemBW)));

        double ideal_time = std::max(dProc,(HtD+DtH));

        std::cout << n*sizeof(float)<< " " << degree << " " << ideal_time << " " << totaltime.count() << " " << (n*(degree+1))/(ideal_time) << " " << ((n*(degree+1))*nbiter)/totaltime.count() << std::endl;

        hipHostFree(array);
        hipHostFree(poly);

        return 0;
    }

