
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define BLOCKSIZE 256

__global__ void polynomial_expansion(float *poly, int degree, int n, float *array)
{
    //TODO: Write code to use the GPU here!
    //code should write the output back to array
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        float temp = array[index];
        float out = 0, xtothepowerof = 1;
        for (int i = 0; i <= degree; i++)
        {
            out += xtothepowerof * poly[i];
            xtothepowerof *= temp;
        }
        array[index] = out;
    }
}

int main(int argc, char *argv[])
{
    //TODO: add usage

    if (argc < 3)
    {
        std::cerr << "usage: " << argv[0] << " n degree" << std::endl;
        return -1;
    }

    char *ptr;
    long long int n = strtol(argv[1],&ptr,10);
    int degree = atoi(argv[2]);
    int nbiter = 2;

    float* array = NULL;
    float* poly = NULL;

    hipHostMalloc((void **)&array,sizeof(float)*n, hipHostMallocDefault);
    hipHostMalloc((void **)&poly,sizeof(float)*(degree+1), hipHostMallocDefault);
    for (int i = 0; i < n; ++i)
        array[i] = 1.;

    for (int i = 0; i < degree + 1; ++i)
        poly[i] = 1.;

        float *d_array, *d_poly;

        hipMalloc((void **)&d_array, n * sizeof(float));
        hipMalloc((void **)&d_poly, (degree + 1) * sizeof(float));
    
        long long int size = n * sizeof(float) / 4;
    
        hipStream_t stream[4];
        for (int i = 0; i < 4; ++i){
            hipStreamCreate(&stream[i]);
        }
    
    
        hipMemcpyAsync(d_poly, poly, (degree + 1) * sizeof(float), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
    
        std::chrono::time_point<std::chrono::system_clock> begin, end;
        begin = std::chrono::system_clock::now();
        for(int k = 1; k <=nbiter; k++){
            for (int i = 0; i < 4; ++i) {
                hipMemcpyAsync(d_array+ i*size, array + i*size,size, hipMemcpyHostToDevice, stream[i]);
                polynomial_expansion <<<((n/4) + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE, 0, stream[i]>>>(d_poly, degree, n/4, d_array + i*size);
                hipMemcpyAsync(array+ i*size, d_array+ i*size,size, hipMemcpyDeviceToHost, stream[i]);
                }
            }
    
        hipDeviceSynchronize();
        end = std::chrono::system_clock::now();
        std::chrono::duration<double> totaltime = (end - begin);
    
        for (int i = 0; i < 4; ++i){
            hipStreamDestroy(stream[i]);
        }
        hipFree(d_array);
        hipFree(d_poly);

        double pciBW = 1.50e+10, gpumemBW = 2.88e+11 , gpuflopRate = 1.73e+12 , pciLat = 8.80597e-06;

        double HtD = pciLat + (((n)*(sizeof(float)))/pciBW);
        double DtH = pciLat + (((n)*(sizeof(float)))/pciBW);

        double dProc = std::max((3.0*(n)*(degree+1)/(gpuflopRate)),((sizeof(float)*(n+degree+1)/(gpumemBW))));
 
        double ideal_time = std::max(dProc,(HtD+DtH));
        
        std::cout << n*sizeof(float)<< " " << array[0]<< " " << degree << " " << totaltime.count() << " " << ((n))/ideal_time << " " << ((n)*nbiter)/totaltime.count() << std::endl;
    
        hipHostFree(array);
        hipHostFree(poly);
    
        return 0;
    }
    